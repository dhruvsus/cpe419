//imports

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
//constants
#define HEIGHT 3000
#define WIDTH 3000
#define NUM_STREAMS 10
#define NUM_BLOCKS 32
#define THREADS_PER_BLOCK 128
#define CACHEAMT 0

__global__ void addMat(int * X, int * Y, int * Z, int numElements, int offset) {
	int myThreadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (myThreadID > offset && myThreadID < offset + numElements) {
		// do the add into Z.
		Z[myThreadID] = X[myThreadID] + Y[myThreadID];
	}
}
int main() {
	int i, *X, *Y, *d_X, *d_Y, *Z, offset, streamSize;
	//clean GPU
	hipDeviceReset();
	//create streams
	hipStream_t streams[NUM_STREAMS];
	for (i = 0; i<NUM_STREAMS; i++){
		hipStreamCreate(&streams[i]);
	}

	//allocate matrix X and Y on host
	hipHostMalloc(&X, HEIGHT * WIDTH * sizeof(int), hipHostMallocDefault);
	hipHostMalloc(&Y, HEIGHT * WIDTH * sizeof(int), hipHostMallocDefault);

	//allocate device versions of X and Y
	hipMalloc(&d_X, HEIGHT * WIDTH * sizeof(int));
	hipMalloc(&d_Y, HEIGHT * WIDTH * sizeof(int));
	//allocate matrix Z on the device
	hipMalloc(&Z, HEIGHT * WIDTH * sizeof(int));

	// for each stream, copy and add
	// copy first part of X and Y
	// this is sequential because on the default stream
	// maybe use 2 streams, async and synchronize the streams
	streamSize = (HEIGHT * WIDTH) / NUM_STREAMS;
	offset = 0;
	// hopefully NUM_STREAMS>=2
	//cudaMemcpyAsync(&d_X[offset], &X[offset], streamSize, cudaMemcpyHostToDevice, streams[0]);
	//cudaMemcpyAsync(&d_Y[offset], &Y[offset], streamSize, cudaMemcpyHostToDevice, streams[1]);
	//cudaStreamSynchronize(streams[0]);
	//cudaStreamSynchronize(streams[1]);
	for(i=0;i<NUM_STREAMS;i++){
		//copy memory
		hipMemcpyAsync(&d_X[offset], &X[offset], streamSize, hipMemcpyHostToDevice, streams[i]);
		hipMemcpyAsync(&d_Y[offset], &Y[offset], streamSize, hipMemcpyHostToDevice, streams[i]);
		addMat<<<NUM_BLOCKS, THREADS_PER_BLOCK, CACHEAMT, streams[i]>>>(d_X, d_Y, Z, streamSize, offset);
	}
	
	//synchronize
	hipDeviceSynchronize();
	for (i = 0; i < NUM_STREAMS; i++){
		hipStreamDestroy( streams[i]);
	}
	hipFree(d_X);
	hipFree(d_Y);
	hipFree(Z);
	hipHostFree(X);
	hipHostFree(Y);
	return 0;
}
