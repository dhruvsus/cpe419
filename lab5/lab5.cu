//imports

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
//constants
#define HEIGHT 3000
#define WIDTH 3000
#define NUM_STREAMS 10
#define NUM_BLOCKS 32
#define THREADS_PER_BLOCK 128
#define CACHEAMT 0

__global__ void addMat(int * X, int * Y, int * Z, int numElements, int offset) {
  int myThreadID = blockIdx.x * blockDim.x + threadIdx.x;
  if (myThreadID > offset && myThreadID < offset + numElements) {
    // do the add into Z.
    Z[myThreadID] = X[myThreadID] + Y[myThreadID];
  }
}
int main() {
  int i, * X, * Y, * d_X, * d_Y, * Z, offset, streamSize;
  //create streams
  hipStream_t streams[NUM_STREAMS];
  for (i = 0; i<NUM_STREAMS; i++){
    hipStreamCreate(&streams[i]);
  }

//allocate matrix X and Y on host
hipHostMalloc((void ** ) & X, HEIGHT * WIDTH * sizeof(int), hipHostMallocDefault);
hipHostMalloc((void ** ) & Y, HEIGHT * WIDTH * sizeof(int), hipHostMallocDefault);

//allocate device versions of X and Y
hipMalloc((void ** ) & d_X, HEIGHT * WIDTH * sizeof(int));
hipMalloc((void ** ) & d_Y, HEIGHT * WIDTH * sizeof(int));
//allocate matrix Z on the device
hipMalloc( & Z, HEIGHT * WIDTH * sizeof(int));

// for each stream, copy and add
// copy first part of X and Y
// this is sequential because on the default stream
// maybe use 2 streams, async and synchronize the streams
streamSize = (HEIGHT * WIDTH) / NUM_STREAMS;
offset = 0;
// hopefully NUM_STREAMS>=2
hipMemcpyAsync( (void *)&d_X[offset], (void *)X[offset], streamSize, hipMemcpyHostToDevice, streams[0]);
hipMemcpyAsync( (void *)&d_Y[offset], (void *)Y[offset], streamSize, hipMemcpyHostToDevice, streams[1]);
hipStreamSynchronize(streams[0]);
hipStreamSynchronize(streams[1]);
i = 0;
while (i < NUM_STREAMS) {
  // do the add async
  addMat <<< NUM_BLOCKS, THREADS_PER_BLOCK, CACHEAMT, streams[i] >>> (d_X, d_Y, Z, streamSize, offset);
  if (i < NUM_BLOCKS - 1) {
    offset = (i + 1) * streamSize;
    // copy next segment async but be careful not to overwrite
    hipMemcpyAsync( (void *)&d_X[offset], (void *)X[offset], streamSize, hipMemcpyHostToDevice, streams[i]);
    hipMemcpyAsync( (void *)&d_Y[offset], (void *)Y[offset], streamSize, hipMemcpyHostToDevice, streams[i]);
    i++;
    hipStreamSynchronize(streams[i]);
  }
}
for (i = 0; i < NUM_STREAMS; i++){
    hipStreamDestroy( streams[i]);
  }
return 0;
}