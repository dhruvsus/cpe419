
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cstdlib>
#include <ctime>

// CUDA kernel to multiply elements of two arrays
#define A_HEIGHT 1000
#define B_WIDTH 1000
#define AB_SHARED 1000

//declare global variables
float* A;
float* B;
float* C;
float* D;
int N;

void matrix_mult_nonthreaded(){
	int i,j,k;
	for(i=0;i<A_HEIGHT;i++){
		for(j=0;j<B_WIDTH;j++){
			for(k=0;k<AB_SHARED;k++){
				D[i*1000+j]+=A[k*1000+j]*B[i*1000+k];
			}
		}
	}
	return;
}

	__global__
void mm(float* A, float* B, float* C, int N)
{
	int i,j,k;
	int threadID = blockDim.x*blockIdx.x+threadIdx.x;
	int gridStride = gridDim.x*blockDim.x;

	for(i=threadID ; i<N; i+=gridStride){
		for(j=0;j<B_WIDTH;j++){
			for(k=0;k<AB_SHARED;k++){
				C[i*1000+j]+=A[k*1000+j]*B[i*1000+k];
			}
		}
	}
}

int main(void)
{
	// Allocate Unified Memory -- accessible from CPU or GPU
	N = A_HEIGHT*AB_SHARED;
	hipMallocManaged(&A, N*sizeof(float));
	hipMallocManaged(&B, N*sizeof(float));
	hipMallocManaged(&C, N*sizeof(float));
	hipMallocManaged(&D, N*sizeof(float));
	float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		A[i] = r;
		B[i] = r;
	}
	
	matrix_mult_nonthreaded();
	// Launch kernel on 4*256 threads

	mm<<<4, 1024>>>(A,B,C,N);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Free memory
	hipFree(A);
	hipFree(B);
	hipFree(C);
	hipFree(D);
	return 0;
}
