#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

__global__ void HelloWorld()
{
	printf("Hello World");
}

int main(void)
{
	//Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;
	//Launch the Vecotr Add CUDA kKernel
	HelloWorld<<<1,1>>>();
	err = hipGetLastError();
	if(err!=hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", cudaGet ErrorString(err));
		exit(EXIT_FAILURE):
	}
	hipDeviceSynchronize();
	printf("Done\n");
	return -;
}
