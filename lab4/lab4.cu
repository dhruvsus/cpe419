#include "hip/hip_runtime.h"
//imports
#include <iostream>
#include <math.h>
#include <cstdlib>
#include <ctime>

//constants for dimensions of matrices
#define A_HEIGHT 1000
#define B_WIDTH 1000
#define AB_SHARED 1000

//declare global variables
float* A;
float* B;
float* C;
int N;

//threaded across cuda enabled GPU for matrix multiplication
__global__ void matrix_mult_threaded(float* A, float* B, float* C, int N)
{
	int i,j,k;
	int threadID = blockDim.x*blockIdx.x+threadIdx.x;
	int gridStride = gridDim.x*blockDim.x;
	//each thread does work[thread] and work[thread+gridStride]
	//until thread+GridStride<N because N >> #threads, so each
	//thread does more work
	//rows of M1
	for(i=threadID ; i<1000; i+=gridStride){
		//columsn of M2
		for(j=0;j<B_WIDTH;j++){
			//columns of M1 = rows of M2
			for(k=0;k<AB_SHARED;k++){
				C[i*1000+j]+=A[i*1000+k]*B[k*1000+i];
			}
		}
	}
}

int main(void)
{	
	//memory allocation
	//host:
	N = A_HEIGHT*AB_SHARED;
	float r;
	int deviceID;
	//GPU specific variables
	hipDeviceProp_t gpuProps;
	//get GPU properties
	hipGetDevice(&deviceID);
	hipGetDeviceProperties(&gpuProps, deviceID);
	int numSM=gpuProps.multiProcessorCount;
	int maxThreadsPerBlock=gpuProps.maxThreadsPerBlock;
	int maxThreadsPerMultiProcessor=gpuProps.maxThreadsPerMultiProcessor;
	//unified:
	hipMallocManaged(&A, N*sizeof(float));
	hipMallocManaged(&B, N*sizeof(float));
	hipMallocManaged(&C, N*sizeof(float));
	
	//prefetch A and B to CPU
	hipMemPrefetchAsync(&A, N*sizeof(float), hipCpuDeviceId);
	hipMemPrefetchAsync(&B, N*sizeof(float), hipCpuDeviceId);


	//Initialize A and B with random values between 0 and 1.0
	for (int i = 0; i < N; i++) {
		r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		A[i] = r;
		B[i] = r;	
	}
		

	//prefetch A, B, and C to GPU
	hipMemPrefetchAsync(&A, N*sizeof(float), deviceID);
	hipMemPrefetchAsync(&B, N*sizeof(float), deviceID);
	hipMemPrefetchAsync(&C, N*sizeof(float), deviceID);

	std::cout<<"SM's "<<numSM<<", maxThreadsPerBlock "<<maxThreadsPerBlock<<", maxThreadsPerMultiProcessor "<<maxThreadsPerMultiProcessor;
	// Launch kernel
	matrix_mult_threaded<<<2*numSM, 128>>>(A,B,C,N);
	
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	
	//fetch C to CPU
	hipMemPrefetchAsync(&C, N*sizeof(float), hipCpuDeviceId);

	// Free memory
	hipFree(A);
	hipFree(B);
	hipFree(C);
	
	return 0;
}
