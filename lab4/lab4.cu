#include "hip/hip_runtime.h"
//imports
#include <iostream>
#include <math.h>
#include <cstdlib>
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//constants for dimensions of matrices
#define A_HEIGHT 1000
#define B_WIDTH 1000
#define AB_SHARED 1000

//init matrix: initialize A and B with value from 0.0 to 1.0
__global__ void init_matrix(float *X, float *Y, int N){
	int i;
	float r;
	int threadID=blockDim.x*blockIdx.x+threadIdx.x;
	int gridStride=gridDim.x*blockDim.x;
	for(i=threadID;i<N;i+=gridStride){
		r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		X[i] = r;
		Y[i] = r;
	}
}

//threaded across cuda enabled GPU for matrix multiplication
__global__ void matrix_mult_threaded(float* A, float* B, float* C, int N)
{
	int i,j,k;
	int threadID = blockDim.x*blockIdx.x+threadIdx.x;
	int gridStride = gridDim.x*blockDim.x;
	//each thread does work[thread] and work[thread+gridStride]
	//until thread+GridStride<N because N >> #threads, so each
	//thread does more work
	//rows of M1
	for(i=threadID ; i<A_HEIGHT; i+=gridStride){
		//columsn of M2
		for(j=0;j<B_WIDTH;j++){
			//columns of M1 = rows of M2
			for(k=0;k<AB_SHARED;k++){
				C[i*AB_SHARED+j]+=A[i*AB_SHARED+k]*B[k*AB_SHARED+i];
			}
		}
	}
}

int main(void)
{	
	//memory allocation
	//host:
	float* A;
	float* B;
	float* C;
	int N;
	N = A_HEIGHT*AB_SHARED;
	float r;
	int deviceID;
	//GPU specific variables
	hipDeviceProp_t gpuProps;
	
	//get GPU properties
	hipGetDevice(&deviceID);
	hipGetDeviceProperties(&gpuProps, deviceID);

	int numSM=gpuProps.multiProcessorCount;
	int maxThreadsPerBlock=gpuProps.maxThreadsPerBlock;
	int maxThreadsPerMultiProcessor=gpuProps.maxThreadsPerMultiProcessor;
	
	//unified:
	hipMallocManaged(&A, N*sizeof(float));
	hipMallocManaged(&B, N*sizeof(float));
	hipMallocManaged(&C, N*sizeof(float));
	
	hiprandCreateGenerator(HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed();
	hiprandState *d_state;
	curdaMalloc(&d, sizerof(hiprandState);
	//initialize A and B on the GPU
	init_matrix<<<2*numSM, 128>>>(A,B,N);

	//prefetch A and B to CPU
	hipMemPrefetchAsync(&A, N*sizeof(float), hipCpuDeviceId);
	hipMemPrefetchAsync(&B, N*sizeof(float), hipCpuDeviceId);

	//prefetch A, B, and C to GPU
	hipMemPrefetchAsync(&A, N*sizeof(float), deviceID);
	hipMemPrefetchAsync(&B, N*sizeof(float), deviceID);
	hipMemPrefetchAsync(&C, N*sizeof(float), deviceID);

	std::cout<<"SM's "<<numSM<<", maxThreadsPerBlock "<<maxThreadsPerBlock<<", maxThreadsPerMultiProcessor "<<maxThreadsPerMultiProcessor;
	// Launch kernel
	matrix_mult_threaded<<<2*numSM, 128>>>(A,B,C,N);
	
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	
	//fetch C to CPU
	hipMemPrefetchAsync(&C, N*sizeof(float), hipCpuDeviceId);

	// Free memory
	hipFree(A);
	hipFree(B);
	hipFree(C);
	
	return 0;
}
