#include "hip/hip_runtime.h"
//imports
#include <iostream>
#include <math.h>
#include <cstdlib>
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <math.h>

//constants for dimensions of matrices
#define A_HEIGHT 8192
#define A_WIDTH 8192
#define THREADSIZE 16
#define THREADSPERBLOCK 128

//init matrix: initialize A and B with value from 0.0 to 1.0
__global__ void initMatrixGPU(float *X, float *Y, int N, hiprandState *state){
	int i, seed=1337;
	int threadID=blockDim.x*blockIdx.x+threadIdx.x;
	int gridStride=gridDim.x*blockDim.x;
	hiprand_init(seed, threadID, 0, &state[threadID]);
	float RANDOM = hiprand_uniform(&state[threadID]);
	for(i=threadID;i<N;i+=gridStride){
		X[i] = RANDOM;
		Y[i] = RANDOM;
	}
}

//matrix addition, non threaded
void matrixAddNonThreaded(float* A, float* B, float* D, int nX, int nY){
	int row, col;
	for (row=0; row<nY; row++){
		for(col=0; col<nX; col++) {
			D[row*nX+col]=A[row*nX+col]+B[row*nX+col];
		}
	}
}

//threaded across cuda enabled GPU for matrix addition
__global__ void matrixAddGPU(float* A, float* B, float* C, int nX, int nY)
{	
	int i, j;
	int xLoc=blockDim.x*blockIdx.x+threadIdx.x;
	int yLoc=blockDim.y*blockIdx.y+threadIdx.y;
	int gridStrideX=blockDim.x*gridDim.x;
	int gridStrideY=blockDim.y*gridDim.y;

	for(i=xLoc;i<nX;i+=gridStrideX){
		for(j=yLoc;j<nY;j+=gridStrideY){
			C[i*nX+j]=A[i*nX+j]+B[i*nX+j];
		}
	}
}

int main(void)
{	
	//memory allocation
	float* A;
	float* B;
	float* C;
	float* D;
	int nX;
	int nY;
	nX=A_WIDTH;
	nY=A_HEIGHT;
	int deviceID;
	int N=nX*nY;
	hiprandState* state;

	// GPU specific variables
	hipDeviceProp_t gpuProps;

	// Get GPU properties
	hipGetDevice(&deviceID);
	hipGetDeviceProperties(&gpuProps, deviceID);
	int numSM=gpuProps.multiProcessorCount;
	int maxThreadsPerBlock=gpuProps.maxThreadsPerBlock;
	int maxThreadsPerMultiProcessor=gpuProps.maxThreadsPerMultiProcessor;
	int maxGridSize=gpuProps.maxGridSize[0];
	int maxThreadsDim=gpuProps.maxThreadsDim[0];	

	const dim3 blockSize(THREADSIZE, THREADSIZE, 1);
	const dim3 gridSize(((A_WIDTH-1)/THREADSIZE)+1,((A_HEIGHT-1)/THREADSIZE)+1);

	// Allocate memory on unified heap and host memory
	hipMallocManaged(&A, nX*nY*sizeof(float));
	hipMallocManaged(&B, nX*nY*sizeof(float));
	hipMemAdvise(&A, N*sizeof(float), hipMemAdviseSetReadMostly, deviceID);
	hipMemAdvise(&B, N*sizeof(float), hipMemAdviseSetReadMostly, deviceID);
	hipMallocManaged(&C, nX*nY*sizeof(float));
	hipMalloc(&state, N*sizeof(hiprandState));

	D = (float*)malloc(N*sizeof(float));

	//current memory status, assuming >Pascal
	//A,B,C allocated on the device
	//nX, nY, deviceID allocated on the host
	//D allocated on the host, as we don't need it on the device.
	//Prefetch A, B, and C onto device
	hipMemPrefetchAsync(&A, N*sizeof(float), deviceID);
	hipMemPrefetchAsync(&B, N*sizeof(float), deviceID);
	hipMemPrefetchAsync(&C, N*sizeof(float), deviceID);

	// Launch init kernel
	initMatrixGPU<<<2*numSM, THREADSPERBLOCK>>>(A,B,nX*nY,state);
	hipDeviceSynchronize();

	// Print GPU info
	std::cout<<"SM's "<<numSM<<", maxThreadsPerBlock "<<maxThreadsPerBlock<<", maxThreadsPerMultiProcessor "<<maxThreadsPerMultiProcessor<<" maxGridSize "<<maxGridSize<<" maxThreadsDim "<<maxThreadsDim<<'\n';
	// Launch add kernel
	matrixAddGPU<<<gridSize, blockSize>>>(A,B,C,nX,nY);
	hipDeviceSynchronize();
	
	// Prefetch A,B to host
	hipMemPrefetchAsync(&A, N*sizeof(float), hipCpuDeviceId);
	hipMemPrefetchAsync(&B, N*sizeof(float), hipCpuDeviceId);
	hipMemPrefetchAsync(&C, N*sizeof(float), hipCpuDeviceId);

	// Sequential matrix addition
	matrixAddNonThreaded(A,B,D,nX,nY);

	//sanity check
	int row, col;
	float dif=0;
	for (row=0; row<nY; row++){
		for(col=0; col<nX; col++)
			dif+=abs(C[row*nX+col]-D[row*nX+col]);
	}
	if(dif < 0.1) printf("SUCCESS\n");
	else printf("FAIL\n");
	printf("%f\n",dif);
	
	// Free memory
	hipFree(A);
	hipFree(B);
	hipFree(C);
	free(D);
	return 0;
}
